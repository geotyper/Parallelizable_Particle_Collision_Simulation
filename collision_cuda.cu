
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <time.h>

#define RAND01 (rand()%2)

typedef enum {
    MODE_PRINT,
    MODE_PERF
} simulation_mode_t;

typedef struct {
    double x;
    double y;
    double vx;
    double vy;
    int colli_p;
    int colli_w;
    double x_n;
    double y_n;
} particle_t;

typedef struct
{
    int pa;
    int pb;
    double time;
} Collision;

__constant__ int n, l, r, s, bnd_far, r_sq_4;
__managed__ particle_t* particles;
__managed__ int *colli_mat, *colli_queue;
__managed__ Collision *colli_time;
__managed__ int count, real_colli;
int host_n, host_l, host_r, host_s, host_bnd_far, host_r_sq_4;
Collision *colli;


/* Current implementation: simplest one 
    @TODO optimize it?
    Every thread-> one particle compare N times
*/

__global__ void find_collisions(int num_threads)
{
    int i = blockIdx.x * num_threads + threadIdx.x;
    if(i>=n) //n threads corresponding to n particles
        return;
    particle_t *P_a, *P_b;
    P_a=particles+i;
    P_a->x_n = P_a->x + P_a->vx;
    P_a->y_n = P_a->y + P_a->vy;
    double lambda, lambda_1, lambda_2, wall_colli, dx1, dy1, Delta, Dx, Dy, dDpdD, DDpDD, dDmdD;
    int cnt;

    lambda_1 = lambda_2 = 2;
    wall_colli = 0;
    if(P_a->x_n<r)
    {
        lambda_1 = (r - P_a->x) / P_a->vx;
        wall_colli = 1;
    }
    else if(P_a->x_n>bnd_far)
    {
        lambda_1 = (bnd_far - P_a->x) / P_a->vx;
        wall_colli = 1;
    }

    if(P_a->y_n<r)
    {
        lambda_2 = (r - P_a->y) / P_a->vy;
        wall_colli = 1;
    }
    else if(P_a->y_n>bnd_far)
    {
        lambda_2 = (bnd_far - P_a->y) / P_a->vy;
        wall_colli = 1;
    }

    if(wall_colli)
    {
        cnt=atomicAdd(&count, 1); // nice!
        colli_time[cnt].pa = i;
        lambda = lambda_1-lambda_2;
        if(lambda==0) // Cornor collision!
        {
            colli_time[cnt].pb = -1; // -1 to present this case.
            colli_time[cnt].time = lambda_1;
        }
        else if(lambda<0) // x wall collision!
        {
            colli_time[cnt].pb = -2; // -2 to present this case.
            colli_time[cnt].time = lambda_1;
        }
        else if(lambda>0) // y wall collision!
        {
            colli_time[cnt].pb = -3; // -3 to present this case.
            colli_time[cnt].time = lambda_2;
        }
    }
    ///////////////
    for(int j=i+1; j<n; j++)
    {
        P_b = particles+j;
        dx1 = P_b->x - P_a->x;
        dy1 = P_b->y - P_a->y;
        // early stop
        Dx = P_b->vx - P_a->vx;
        Dy = P_b->vy - P_a->vy;
        dDpdD = dx1*Dx + dy1*Dy;
        if(dDpdD>=0) // To judge the right direction
            continue;
        // Case 2: overlap at startup:
        ////////////////
        Delta = dx1*dx1 + dy1*dy1;
        if(Delta - r_sq_4<=0 && Delta!=0)
        {
            cnt=atomicAdd(&count, 1);
            colli_time[cnt].time = 0;
            colli_time[cnt].pa = i;
            colli_time[cnt].pb = j; // pa always smaller than pb
            continue; // no need to further detect.
        }
        ////////////////
        // Case 3: Normal collision case
        ////////////////
        DDpDD = Dx*Dx + Dy*Dy;
        dDmdD = dx1*Dy - dy1*Dx;
        Delta = r_sq_4*DDpDD - dDmdD*dDmdD;
        if(Delta<=0)
            continue;
        Delta = sqrtf(Delta);
        lambda = (-dDpdD - Delta)/DDpDD;
        if(lambda<1)
        {
            cnt=atomicAdd(&count, 1);
            colli_time[cnt].time = lambda;
            colli_time[cnt].pa = i;
            colli_time[cnt].pb = j;
        }
        ////////////////
    }
}

__host__ void find_real_collisions()
{
    for(int i=0;i<count;i++)
    {
        colli = colli_time+i;
        if(!colli_mat[colli->pa])
        {
            if(colli->pb<0)
            {
                colli_mat[colli->pa] = 1;
                colli_queue[real_colli++] = i;
                particles[colli->pa].colli_w++;
            }
            else if(!colli_mat[colli->pb])
            {
                colli_mat[colli->pa] = 1;
                colli_mat[colli->pb] = 1;
                colli_queue[real_colli++] = i;
                particles[colli->pa].colli_p++;
                particles[colli->pb].colli_p++;
            }
        }
    }
}

__device__ void bound_pos(particle_t *p)
{
    double tx=0,ty=0;
    if(p->x_n>bnd_far)
        tx = (p->x_n-bnd_far)/p->vx;
    else if(p->x_n<r)
        tx = (p->x_n-r)/p->vx;
    if(p->y_n>bnd_far)
        ty = (p->y_n-bnd_far)/p->vy;
    else if(p->y_n<r)
        ty = (p->y_n-r)/p->vy;

    tx =ty = tx>ty?tx:ty;
    p->x_n = p->x_n - tx*p->vx;
    p->y_n = p->y_n - ty*p->vy;
}

__global__ void update_particle(int num_threads)
{
    int i = blockIdx.x * num_threads + threadIdx.x;
    if(i>=real_colli) //n threads corresponding to n collisions
        return;
    particle_t *P_a, *P_b;
    Collision *Colli;
    double Dx ,Dy, Delta, dx1, dy1, dx2, dy2, DDpDD;
    Colli = colli_time + colli_queue[i];
    if(Colli->pb==-1) // Cornor colli;
    {
        P_a = particles + Colli->pa;
        P_a->vx = -1*P_a->vx;
        P_a->vy = -1*P_a->vy;
        P_a->x_n = P_a->x+(1-2*Colli->time)*P_a->vx;
        P_a->y_n = P_a->y+(1-2*Colli->time)*P_a->vy;
        bound_pos(P_a);
    }
    else if(Colli->pb==-2)//  X wall colli;
    {
        P_a = particles + Colli->pa;
        P_a->vx = -1*P_a->vx;
        P_a->x_n = P_a->x+(1-2*Colli->time)*P_a->vx;
        bound_pos(P_a);
    }
    else if(Colli->pb==-3)// Y wall colli;
    {
        P_a = particles + Colli->pa;
        P_a->vy = -1*P_a->vy;
        P_a->y_n = P_a->y+(1-2*Colli->time)*P_a->vy;
        bound_pos(P_a);
    }
    else // P-P colli;
    {
        P_a = particles + Colli->pa;
        P_b = particles + Colli->pb;
        P_a->x_n = P_a->x + Colli->time*P_a->vx;
        P_a->y_n = P_a->y + Colli->time*P_a->vy;
        P_b->x_n = P_b->x + Colli->time*P_b->vx;
        P_b->y_n = P_b->y + Colli->time*P_b->vy;
        Dx = P_b->x_n - P_a->x_n;
        Dy = P_b->y_n - P_a->y_n;
        Delta = 1 - Colli->time;
        /* To reduce var:
         dx1: nv1; dy1: tv1;
         dx2: nv2; dy2: tv2;
        */
        dx1 = Dx*P_a->vx + Dy*P_a->vy;
        dy1 = Dx*P_a->vy - Dy*P_a->vx;
        dx2 = Dx*P_b->vx + Dy*P_b->vy;
        dy2 = Dx*P_b->vy - Dy*P_b->vx;
        DDpDD = Dx*Dx + Dy*Dy;
        if(DDpDD!=0)
        {
            // Update velocities
            P_a->vx = (dx2*Dx-dy1*Dy)/DDpDD;
            P_a->vy = (dx2*Dy+dy1*Dx)/DDpDD;
            P_b->vx = (dx1*Dx-dy2*Dy)/DDpDD;
            P_b->vy = (dx1*Dy+dy2*Dx)/DDpDD;
        }
        // Update position
        P_a->x_n = P_a->x_n + Delta*P_a->vx;
        P_a->y_n = P_a->y_n + Delta*P_a->vy;
        bound_pos(P_a);
        P_b->x_n = P_b->x_n + Delta*P_b->vx;
        P_b->y_n = P_b->y_n + Delta*P_b->vy;
        bound_pos(P_b);
    }
}

__host__ double doubleRand(double min, double max) // return [min, max] double vars
{
    return min+(max-min)*(rand() / (double)RAND_MAX);
}

__host__ void randomise_particles()
{
    /* Implement randomisation */
    for(int i=0; i<host_n; i++)
    {
        particles[i].x = doubleRand(host_r,host_bnd_far);
        particles[i].y = doubleRand(host_r,host_bnd_far);
        particles[i].vx = (1 - 2*RAND01)*doubleRand(host_l/(double)8.0/host_r,host_l/(double)4.0);
        particles[i].vy = (1 - 2*RAND01)*doubleRand(host_l/(double)8.0/host_r,host_l/(double)4.0);
    }
}

__host__ void print_particles(int step)
{
    int i;
    for (i = 0; i < host_n; i++) {
        printf("%d %d %10.8lf %10.8lf %10.8lf %10.8lf\n", step, i, particles[i].x, particles[i].y,
            particles[i].vx, particles[i].vy);
    }
}

__host__ void print_statistics(int num_step)
{
    int i;
    for (i = 0; i < host_n; i++) {
        printf("%d %d %10.8lf %10.8lf %10.8lf %10.8lf %d %d\n", num_step, i, particles[i].x,
            particles[i].y, particles[i].vx, particles[i].vy,
            particles[i].colli_p, particles[i].colli_w);
    }
}

__host__ int compare (const void * a, const void * b)
{
    Collision *colli_A = (Collision*)a;
    Collision *colli_B = (Collision*)b;
    double cmpf = colli_A->time - colli_B->time;
    if(cmpf!=0)
        return cmpf<0?-1:1;
    else
    {
        int cmpt = colli_A->pa - colli_B->pa;
        if(cmpt!=0)
            return cmpt;
        else
            return colli_A->pb - colli_B->pb;
    }
}

__host__ void check_cuda_errors()
{
    hipError_t rc;
    rc = hipGetLastError();
    if (rc != hipSuccess)
    {
        printf("Last CUDA error %s\n", hipGetErrorString(rc));
    }
}

int main(int argc, char** argv)
{
    int i,j=0;
    double x, y, vx, vy;
    int num_blocks, num_threads;
    int step;
    simulation_mode_t mode;
    char mode_buf[6];

    freopen("./inputs.txt","r",stdin);
    freopen("./outputs.txt","w",stdout);
    srand(0);
    if (argc != 3) {
        printf("Usage:\n%s num_blocks num_threads\n", argv[0]);
        return 1;
    }

    num_blocks = atoi(argv[1]);
    num_threads = atoi(argv[2]);

    scanf("%d", &host_n);
    scanf("%d", &host_l);
    scanf("%d", &host_r);
    scanf("%d", &host_s);
    scanf("%5s", mode_buf);
    host_bnd_far = host_l - host_r;
    host_r_sq_4 = host_r * host_r * 4;

    hipMallocManaged((void**)&particles, sizeof(particle_t) * host_n);
    hipMallocManaged((void**)&colli_mat,sizeof(int) * host_n);
    hipMallocManaged((void**)&colli_queue,sizeof(int) * host_n);
    hipMallocManaged((void**)&colli_time, sizeof(Collision) * host_n*(host_n+1)/2);

    while (scanf("%d %lf %lf %lf %lf", &i, &x, &y, &vx, &vy) != EOF) {
        j++;
        particles[i].x = x;
        particles[i].y = y;
        particles[i].vx = vx;
        particles[i].vy = vy;
        particles[i].colli_p = 0;
        particles[i].colli_w = 0;
    }
    if (j==0) {
        randomise_particles();
    }
    else if(j!=host_n){
        fprintf(stderr, "Not enough particle parameters!\n");
        exit(1);
    }

    mode = strcmp(mode_buf, "print") == 0 ? MODE_PRINT : MODE_PERF;

    /* Copy to GPU constant memory */
    hipMemcpyToSymbol(HIP_SYMBOL(n), &host_n, sizeof(n));
    hipMemcpyToSymbol(HIP_SYMBOL(l), &host_l, sizeof(l));
    hipMemcpyToSymbol(HIP_SYMBOL(r), &host_r, sizeof(r));
    hipMemcpyToSymbol(HIP_SYMBOL(s), &host_s, sizeof(s));
    hipMemcpyToSymbol(HIP_SYMBOL(bnd_far), &host_bnd_far, sizeof(bnd_far));
    hipMemcpyToSymbol(HIP_SYMBOL(r_sq_4), &host_r_sq_4, sizeof(r_sq_4));
    check_cuda_errors();

    for (step = 0; step < host_s; step++) {
        if (step == 0) {
            print_particles(step);
        }
        count=0; //initialize collision numbers every step
        real_colli=0;
        hipMemset(colli_mat, 0, sizeof(int) * host_n);
        /* Call the kernel */
        find_collisions<<<num_blocks, num_threads>>>(num_threads); //one particle one thread
        /* Barrier */
        hipDeviceSynchronize();
        // find real collisions
        qsort(colli_time, count, sizeof(Collision), compare);
        find_real_collisions();
        /* Call the kernel */
        num_threads=(real_colli-1)/num_blocks+1;
        update_particle<<<num_blocks, num_threads>>>(num_threads); //one collision one thread
        /* Barrier */
        hipDeviceSynchronize();
        for(i=0;i<host_n;i++)
        {
            particles[i].x = particles[i].x_n;
            particles[i].y = particles[i].y_n;
        }
    }

    print_statistics(host_s);

    fclose(stdin);
    fclose(stdout);
    hipFree(particles);
    hipFree(colli_time);
    hipFree(colli_mat);
    hipFree(colli_queue);

    return 0;
}
